
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define SEC_AS_NANO 1000000000.0

struct _matriz
{
	int n;
	int m;
	int **cont;
}; typedef struct _matriz Matriz;

Matriz *criarMatriz(int n, int m)
{
	Matriz *mat = (Matriz*) malloc(sizeof(Matriz));

	mat->n = n;
	mat->m = m;
	mat->cont = (int**) malloc(n * sizeof(int*));
	for(int i = 0; i < n; i++)
		mat->cont[i] = (int*) malloc(m * sizeof(int));

	return mat;
}

void liberarMatriz(Matriz *m)
{
	for(int i = 0; i < m->n; i++)
		free(m->cont[i]);
	free(m->cont);
	free(m);
}

Matriz *gerarMatriz(int n, int m)
{
	Matriz *mat = criarMatriz(n, m);
	
	for(int i = 0; i < n; i++)
		for(int j = 0; j < m; j++)
			{
				mat->cont[i][j] = rand() % 100;
			}

	return mat;
}

void printarMatriz(Matriz *mat)
{
	for(int i = 0; i < mat->n; i++)
	{
		for(int j = 0; j < mat->m; j++)
			printf("%d ", mat->cont[i][j]);
		printf("\n");
	}
}

void multiplicarMatrizes(Matriz *a, Matriz *b, Matriz *c)
{
	for(int i = 0; i < a->n; i++)
		for(int j = 0; j < b->m; j++)
		{
			c->cont[i][j] = 0;
			for(int k = 0; k < a->m; k++)
				c->cont[i][j] += a->cont[i][k] * b->cont[k][j];
		}
}

Matriz *lerMatriz(char *nome, int n, int m)
{
	Matriz *mat = criarMatriz(n, m);

	FILE *f = fopen(nome, "r");

	for(int i = 0; i < n; i++)
		for(int j = 0; j < m; j++)
			fscanf(f, " %d", &(mat->cont[i][j]));

	fclose(f);

	return mat;
}

void salvarMatriz(Matriz *mat)
{
	static int i = 0;

	char nome[100];
	sprintf(nome, "%d-%dx%d.txt", i, mat->n, mat->m);

	FILE *f = fopen(nome, "w");

	for(int i = 0; i < mat->n; i++)
	{
		for(int j = 0; j < mat->m; j++)
			fprintf(f, "%d ", mat->cont[i][j]);
		fprintf(f, "\n");
	}

	fclose(f);
	i++;
}

struct _input
{
	Matriz *a;
	Matriz *b;
	Matriz *c;
	short int salvar;
}; typedef struct _input Input;

Input *lerInput(int argc, char **argv)
{
	if(argc >= 6)
	{
		Input *i = (Input *) malloc(sizeof(Input));
		i->salvar = 0;

		int n1, m1, n2, m2;
		char op;

		op = argv[1][0];
		
		sscanf(argv[2], " %d", &n1);
		sscanf(argv[3], " %d", &m1);
		sscanf(argv[4], " %d", &n2);
		sscanf(argv[5], " %d", &m2);
	
		if(m1 == n2)
		{
			Matriz *a, *b, *c;		
			
			switch(op)
			{
				case 'g':
					srand(time(NULL));
					a = gerarMatriz(n1, m1);
					b = gerarMatriz(n2, m2);
					if(argc == 7 && argv[6][0] == 's')
						i->salvar = 1;
					break;
				case 'f':
					a = lerMatriz(argv[6], n1, m1);
					b = lerMatriz(argv[7], n2, m2);
					break;
				default:
					return 0;
			}
			c = criarMatriz(n1, m2);

			i->a = a;
			i->b = b;
			i->c = c;

			return i;
		}
		else
			printf("Matrizes Incompativeis!\n");
	}
	else
		printf("Argumentos invalidos!\n");

	return NULL;
}

double medirTempoInput(Input **i, int argc, char **argv, Input *ler(int, char**))
{
	timespec ini, fim;
	clock_gettime(CLOCK_REALTIME, &ini);
	*i = ler(argc, argv);
	clock_gettime(CLOCK_REALTIME, &fim);

	double iniSec = ini.tv_sec + ini.tv_nsec / SEC_AS_NANO;
	double fimSec = fim.tv_sec + fim.tv_nsec / SEC_AS_NANO;	

	return (fimSec - iniSec);
}

double medirTempoExecMul(Input *i)
{
	timespec ini, fim;
	clock_gettime(CLOCK_REALTIME, &ini);
	multiplicarMatrizes(i->a, i->b, i->c);
	clock_gettime(CLOCK_REALTIME, &fim);

	double iniSec = ini.tv_sec + ini.tv_nsec / SEC_AS_NANO;
	double fimSec = fim.tv_sec + fim.tv_nsec / SEC_AS_NANO;	

	return (fimSec - iniSec);
}

void salvarELiberarMatrizes(Input *i)
{	
	if(i->salvar)
	{
		salvarMatriz(i->a);
		salvarMatriz(i->b);
	}
	salvarMatriz(i->c);

	liberarMatriz(i->a);
	liberarMatriz(i->b);
	liberarMatriz(i->c);
	free(i);
}

int verificarArgumentos(int argc, char **argv)
{
	if(argc < 6)
	{
		printf("Poucos argumentos\n"
			"#  FONTE: f para arquivos, g para gerar\n"
			"#  LINSA: linhas para matriz A\n"
			"#  COLSA: colunas para matriz A\n"
			"#  LINSB: linhas para matriz B\n"
			"#  COLSB: colunas para matriz B\n"
			"#  ARQA: arquivo com a matriz A\n"
			"#  ARQB: arquivo com a matriz B\n"
			"#  SAV (opcional): salva as matrizes A e B geradas"
			"##  ./prog f LA CA LB CB ARQA ARQB\n"
			"##  ./prog g LA CA LB CB SAV\n");
		return 0;
	}
	else
	{
		if(argv[1][0] != 'f' && argv[1][0] != 'g')
		{
			printf("Argumento fonte invalido, use g ou f\n");
			return 0;
		}

		int aux;
		for(int i = 2; i < 6; i++)
			if(!sscanf(argv[i], "%d", &aux))
			{
				printf("O valor %d nao e um numero, informe as dimensoes das matrizes A e B\n", (i - 1));
				return 0;
			}

		if(argv[1][0] == 'g')
			if(argc == 7)
				if(argv[6][0] != 's')
				{
					printf("Adicione s para salvar as matrizes A e B\n");
					return 0;
				}

		if(argv[1][0] == 'f')
		{
			FILE *f;
			if((f = fopen(argv[6], "r")) == NULL)
			{
				printf("O arquivo da matriz A nao existe\n");
				return 0;
			}
			else
				fclose(f);
			if((f = fopen(argv[7], "r")) == NULL)
			{
				printf("O arquivo da matriz B nao existe\n");
				return 0;
			}
			else
				fclose(f);
		}
		
	}

	return 1;
}

int main(int argc, char ** argv)
{
	if(verificarArgumentos(argc, argv))
	{
		Input *i;
		printf("Tempo de criacao: %lf\n", medirTempoInput(&i, argc, argv, &lerInput));
		printf("Tempo de execucao: %lf\n", medirTempoExecMul(i));
		salvarELiberarMatrizes(i);
	}

	return 0;
}
