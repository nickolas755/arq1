/*
#v1
Ideia: Transformar as matrizes em transpostas para nao precisar fazer ler dois ponteiros, apenas usar o deslocamento
Resultado: Aumento de performance. Tempo 1/8 vezes o anterior #8.2 -> 1.1

#v2
Ideia: Transformar matriz em vetor para preparar para CUDA
Resultado: Perda de desempenho. Tempo 2.4 vezes o anterior #1.1 -> 2.4

#v2.1
Ideia: otimizar o codigo antes do CUDA procurando por calculos repetidos e os atribuindo a auxiliares
Resultado: Ganho de desempenho. Tempo 10/15 vezes o anterior #2.4 -> 1.55
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NTRANS 0
#define TRANS 1

struct _matriz
{
	int n;
	int m;
	int *cont;
}; typedef struct _matriz Matriz;

Matriz *criarMatriz(int n, int m)
{
	Matriz *mat = (Matriz*) malloc(sizeof(Matriz));

	mat->n = n;
	mat->m = m;
	mat->cont = (int*) malloc(n * m * sizeof(int*));

	return mat;
}

void liberarMatriz(Matriz *m)
{
	free(m->cont);
	free(m);
}

Matriz *gerarMatriz(int n, int m)
{
	Matriz *mat = criarMatriz(n, m);
	
	for(int i = 0; i < n; i++)
		for(int j = 0; j < m; j++)
			{
				mat->cont[i * m + j] = rand() % 100;
			}

	return mat;
}

void printarMatriz(Matriz *mat)
{
	for(int i = 0; i < mat->n; i++)
	{
		for(int j = 0; j < mat->m; j++)
			printf("%d ", mat->cont[i * mat->m + j]);
		printf("\n");
	}
}

void multiplicarMatrizes(Matriz *a, Matriz *b, Matriz *c)
{
	int aux;
	for(int i = 0; i < a->n; i++)
		for(int j = 0; j < b->n; j++)
		{
			aux = i * c->m + j;
			c->cont[aux] = 0;
			for(int k = 0; k < b->m; k++)
				c->cont[aux] += a->cont[i * a->m + k] * b->cont[j * b->m + k];
		}
}

Matriz *lerMatriz(char *nome, int n, int m, short int trans)
{
	Matriz *mat = NULL;
	FILE *f = fopen(nome, "r");
	if(trans)
	{
		mat = criarMatriz(m, n);

		for(int i = 0; i < n; i++)
			for(int j = 0; j < m; j++)
				fscanf(f, " %d", &(mat->cont[j * n + i]));
	}
	else
	{
		mat = criarMatriz(n, m);

		for(int i = 0; i < n; i++)
			for(int j = 0; j < m; j++)
				fscanf(f, " %d", &(mat->cont[i * m + j]));
	}
	fclose(f);

	return mat;
}

void salvarMatriz(Matriz *mat, short int trans)
{
	static int i = 0;
	char nome[100];

	if(trans)	sprintf(nome, "%d-%dx%d.txt", i, mat->m, mat->n);
	else		sprintf(nome, "%d-%dx%d.txt", i, mat->n, mat->m);

	FILE *f = fopen(nome, "w");

	if(trans)
		for(int i = 0; i < mat->m; i++)
		{
			for(int j = 0; j < mat->n; j++)
				fprintf(f, "%d ", mat->cont[j * mat->m + i]);
			fprintf(f, "\n");
		}
	else
		for(int i = 0; i < mat->n; i++)
		{
			for(int j = 0; j < mat->m; j++)
				fprintf(f, "%d ", mat->cont[i * mat->m + j]);
			fprintf(f, "\n");
		}

	fclose(f);
	i++;
}

struct _input
{
	Matriz *a;
	Matriz *b;
	Matriz *c;
	short int salvar;
}; typedef struct _input Input;

Input *lerInput(int argc, char **argv)
{
	if(argc >= 6)
	{
		Input *i = (Input *) malloc(sizeof(Input));
		i->salvar = 0;

		int n1, m1, n2, m2;
		char op;

		op = argv[1][0];
		
		sscanf(argv[2], " %d", &n1);
		sscanf(argv[3], " %d", &m1);
		sscanf(argv[4], " %d", &n2);
		sscanf(argv[5], " %d", &m2);
	
		if(m1 == n2)
		{
			Matriz *a, *b, *c;		
			
			switch(op)
			{
				case 'g':
					srand(time(NULL));
					a = gerarMatriz(n1, m1);
					b = gerarMatriz(m2, n2); //INVERTIDOS PARA A TRANSPOSTA
					if(argc == 7 && argv[6][0] == 's')
						i->salvar = 1;
					break;
				case 'f':
					a = lerMatriz(argv[6], n1, m1, NTRANS);
					b = lerMatriz(argv[7], n2, m2, TRANS);
					break;
				default:
					return 0;
			}
			c = criarMatriz(n1, m2);

			i->a = a;
			i->b = b;
			i->c = c;

			return i;
		}
		else
			printf("Matrizes Incompativeis!\n");
	}
	else
		printf("Argumentos invalidos!\n");

	return NULL;
}

double medirTempoExecMul(Input *i)
{
	clock_t tempo = clock();
	multiplicarMatrizes(i->a, i->b, i->c);
	tempo = clock() - tempo;

	return ((double) tempo / CLOCKS_PER_SEC);
}

void salvarELiberarMatrizes(Input *i)
{	
	if(i->salvar)
	{
		salvarMatriz(i->a, NTRANS);
		salvarMatriz(i->b, TRANS);
	}
	salvarMatriz(i->c, NTRANS);

	liberarMatriz(i->a);
	liberarMatriz(i->b);
	liberarMatriz(i->c);
	free(i);
}

int main(int argc, char ** argv)
{
	clock_t tempo = clock();
	Input *i = lerInput(argc, argv);
	printf("Tempo de criacao: %lf\n", (((double) clock() - tempo) / CLOCKS_PER_SEC));
	printf("Tempo de execucao: %lf\n", medirTempoExecMul(i));
	salvarELiberarMatrizes(i);

	return 0;
}
