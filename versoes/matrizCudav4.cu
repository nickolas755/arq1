/*
#v1
Ideia: Transformar as matrizes em transpostas para nao precisar fazer ler dois ponteiros, apenas usar o deslocamento
Resultado: Aumento de performance. Tempo 1/8 vezes o anterior #8.2 -> 1.1

#v2
Ideia: Transformar matriz em vetor para preparar para CUDA
Resultado: Perda de desempenho. Tempo 2.4 vezes o anterior #1.1 -> 2.4
	#v2.1
	Ideia: otimizar o codigo antes do CUDA procurando por calculos repetidos e os atribuindo a auxiliares
	Resultado: Ganho de desempenho. Tempo 10/15 vezes o anterior #2.4 -> 1.55

#v3
Ideia: Utilizar a GPU para fazer os calculos quando a matriz for grande
Resultado: Ganho de desempenho. Tempo 10/50 vezes o anterior #1.55 -> 0.35

#v4
Otimizar processamento na CPU com e SSE
Resultado: N/A
	#v4.1
	Utilizar AVX ao inves de SSE
	Resultado: Ganho de desempenho. Tempo 10/28 vezes o anterior # 1.55 -> 0.55
		#v4.2
		Encontradas novas contas frequentes e foram trocadas para variaveis auxiliar
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <immintrin.h>

#define NTRANS 0
#define TRANS 1

#define OPMAX 1024
#define MAXTHREADS_GPU 32	//32*32=1024, Nao se deve ter mais que 1024 threads por bloco

#define MAXTHREADS_CPU 125673	//Utilizado cat /proc/sys/kernel/threads-max para encontrar

#define AVXJUMP 8

struct _matriz
{
	int n;
	int m;
	int *cont;
}; typedef struct _matriz Matriz;

struct _input
{
	Matriz *a;
	Matriz *b;
	Matriz *c;
	short int salvar;
}; typedef struct _input Input;

Matriz *criarMatriz(int n, int m)
{
	Matriz *mat = (Matriz*) malloc(sizeof(Matriz));

	mat->n = n;
	mat->m = m;
	mat->cont = (int*) malloc(n * m * sizeof(int*));

	return mat;
}

void liberarMatriz(Matriz *m)
{
	free(m->cont);
	free(m);
}

Matriz *gerarMatriz(int n, int m)
{
	Matriz *mat = criarMatriz(n, m);
	
	for(int i = 0; i < n; i++)
		for(int j = 0; j < m; j++)
			{
				mat->cont[i * m + j] = rand() % 100;
			}

	return mat;
}

void printarMatriz(Matriz *mat)
{
	for(int i = 0; i < mat->n; i++)
	{
		for(int j = 0; j < mat->m; j++)
			printf("%d ", mat->cont[i * mat->m + j]);
		printf("\n");
	}
}

/*void multiplicarMatrizes(Matriz *a, Matriz *b, Matriz *c)
{
	int aux;
	for(int i = 0; i < a->n; i++)
		for(int j = 0; j < b->n; j++)
		{
			aux = i * c->m + j;
			c->cont[aux] = 0;
			for(int k = 0; k < b->m; k++)
				c->cont[aux] += a->cont[i * a->m + k] * b->cont[j * b->m + k];
		}
}*/

void multiplicarMatrizesAVX(Matriz *matA, Matriz *matB, Matriz *matC)
{
	int *a = matA->cont, *b = matB->cont, *c = matC->cont;
	int aN = matA->n, bN = matB->n, aM = matA->m, bM = matB->m, cM = matC->m, k;

	__m256i mask = _mm256_setr_epi32(-1, -2, -3, -4, -5, -6, -7, -8);	//MASCARA INFORMA QUE SERAO USADOS OS 256 BITS DO AVX
	__m256i regMults;	//AUXILIAR, GUARDA O VETOR DE 8 MULTIPLICACOES

	int indiceC, indiceB, indiceA;
	int limK = bM - (bM % AVXJUMP);
	for(int i = 0; i < aN; i++)
	{
		indiceA = i * aM;
		for(int j = 0; j < bN; j++)
		{
			indiceB = j * bM;
			indiceC = i * cM + j;
			c[indiceC] = 0;
			for(k = 0; k < limK; k += AVXJUMP)	//LOOP PARA MULTIPLOS DE 8, LIMITE DO AVX
			{
				regMults = _mm256_mullo_epi32(_mm256_maskload_epi32(&a[indiceA + k], mask),
								_mm256_maskload_epi32(&b[indiceB + k], mask));
				regMults = _mm256_hadd_epi32(regMults, regMults);
				regMults = _mm256_hadd_epi32(regMults, regMults);
				c[indiceC] += _mm256_extract_epi32(regMults, 0) + _mm256_extract_epi32(regMults, 7);
			}
		}
	}

	if(limK < bM)	//SEPARADO DO LOOP J PRINCIPAL PARA EVITAR CALCULOS EM MULTIPLOS DE 8
	{
		for(int i = 0; i < aN; i++)
		{
			indiceA = i * aM;
			for(int j = 0; j < bN; j++)
			{
				indiceB = j * bM;
				indiceC = i * cM + j;
				for(k = limK; k < bM; k++)	//LOOP PARA CASO O N°COL DE C NAO SEJA MULTIPLO DE 8
					c[indiceC] += a[indiceA + k] * b[indiceB + k];
			}
		}

	}
}

//KERNEL QUE EXECUTA AS INSTRUCOES NA GPU
__global__ void kernelMulMatriz(int *a, int nA, int mA, int *b, int nB, int mB, int *c, int mC)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if(i < nA && j < nB)
	{
		int aux = i * mC + j;
		c[aux] = 0;
		for(int k = 0; k < mB; k++)
			c[aux] += a[i * mA + k] * b[j * mB + k];
	}
}

//FUNCAO QUE PREPARA PARA A GPU
void multiplicarMatrizesCUDA(Matriz *a, Matriz *b, Matriz *c)
{
	int *d_a, *d_b, *d_c, opCount = a->n * b->n;
	dim3 blocksPerGrid(1, 1), threadsPerBlock(a->n, b->n);
	if(opCount > OPMAX)
	{
		threadsPerBlock.x = MAXTHREADS_GPU;
		threadsPerBlock.y = MAXTHREADS_GPU;
		blocksPerGrid.x = ceil(((double) a->n / MAXTHREADS_GPU));
		blocksPerGrid.y = ceil(((double) b->n / MAXTHREADS_GPU));
	}

	hipMalloc(&d_a, sizeof(int) * a->n * a->m);
	hipMalloc(&d_b, sizeof(int) * b->n * b->m);
	hipMalloc(&d_c, sizeof(int) * c->n * c->m);
	hipMemcpy(d_a, a->cont, sizeof(int) * a->n * a->m, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b->cont, sizeof(int) * b->n * b->m, hipMemcpyHostToDevice);

	kernelMulMatriz<<<blocksPerGrid, threadsPerBlock>>> (d_a, a->n, a->m, d_b, b->n, b->m, d_c, c->m);

	hipMemcpy(c->cont, d_c, sizeof(int) * c->n * c->m, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	hipDeviceSynchronize();
}

Matriz *lerMatriz(char *nome, int n, int m, short int trans)
{
	Matriz *mat = NULL;
	FILE *f = fopen(nome, "r");
	if(trans)
	{
		mat = criarMatriz(m, n);

		for(int i = 0; i < n; i++)
			for(int j = 0; j < m; j++)
				fscanf(f, " %d", &(mat->cont[j * n + i]));
	}
	else
	{
		mat = criarMatriz(n, m);

		for(int i = 0; i < n; i++)
			for(int j = 0; j < m; j++)
				fscanf(f, " %d", &(mat->cont[i * m + j]));
	}
	fclose(f);

	return mat;
}

void salvarMatriz(Matriz *mat, short int trans)
{
	static int i = 0;
	char nome[100];

	if(trans)	sprintf(nome, "%d-%dx%d.txt", i, mat->m, mat->n);
	else		sprintf(nome, "%d-%dx%d.txt", i, mat->n, mat->m);

	FILE *f = fopen(nome, "w");

	if(trans)
		for(int i = 0; i < mat->m; i++)
		{
			for(int j = 0; j < mat->n; j++)
				fprintf(f, "%d ", mat->cont[j * mat->m + i]);
			fprintf(f, "\n");
		}
	else
		for(int i = 0; i < mat->n; i++)
		{
			for(int j = 0; j < mat->m; j++)
				fprintf(f, "%d ", mat->cont[i * mat->m + j]);
			fprintf(f, "\n");
		}

	fclose(f);
	i++;
}

Input *lerInput(int argc, char **argv)
{
	if(argc >= 6)
	{
		Input *i = (Input *) malloc(sizeof(Input));
		i->salvar = 0;

		int n1, m1, n2, m2;
		char op;

		op = argv[1][0];
		
		sscanf(argv[2], " %d", &n1);
		sscanf(argv[3], " %d", &m1);
		sscanf(argv[4], " %d", &n2);
		sscanf(argv[5], " %d", &m2);
	
		if(m1 == n2)
		{
			Matriz *a, *b, *c;		
			
			switch(op)
			{
				case 'g':
					srand(time(NULL));
					a = gerarMatriz(n1, m1);
					b = gerarMatriz(m2, n2); //INVERTIDOS PARA A TRANSPOSTA
					if(argc == 7 && argv[6][0] == 's')
						i->salvar = 1;
					break;
				case 'f':
					a = lerMatriz(argv[6], n1, m1, NTRANS);
					b = lerMatriz(argv[7], n2, m2, TRANS);
					break;
				default:
					return 0;
			}
			c = criarMatriz(n1, m2);

			i->a = a;
			i->b = b;
			i->c = c;

			return i;
		}
		else
			printf("Matrizes Incompativeis!\n");
	}
	else
		printf("Argumentos invalidos!\n");

	return NULL;
}

double medirTempoExecMul(Input *i, void mul(Matriz*, Matriz*, Matriz*))
{
	clock_t tempo = clock();
	mul(i->a, i->b, i->c);
	tempo = clock() - tempo;

	return ((double) tempo / CLOCKS_PER_SEC);
}

void salvarELiberarMatrizes(Input *i)
{	
	if(i->salvar)
	{
		salvarMatriz(i->a, NTRANS);
		salvarMatriz(i->b, TRANS);
	}
	salvarMatriz(i->c, NTRANS);

	liberarMatriz(i->a);
	liberarMatriz(i->b);
	liberarMatriz(i->c);
	free(i);
}

int main(int argc, char ** argv)
{
	clock_t tempo = clock();
	Input *i = lerInput(argc, argv);
	printf("Tempo de criacao: %lf\n", (((double) clock() - tempo) / CLOCKS_PER_SEC));
	printf("Tempo de execucao: %lf\n", medirTempoExecMul(i, &multiplicarMatrizesAVX));
	salvarELiberarMatrizes(i);

	return 0;
}
