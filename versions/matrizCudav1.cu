/*
#v1
Ideia: Transformar as matrizes em transpostas para nao precisar fazer ler dois ponteiros, apenas usar o deslocamento
Resultado: Aumento de performance para mais de 8 vezes

#v2
Ideia: Calcular na GPU
Resultado: N/A
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NTRANS 0
#define TRANS 1

struct _matriz
{
	int n;
	int m;
	int **cont;
}; typedef struct _matriz Matriz;

Matriz *criarMatriz(int n, int m)
{
	Matriz *mat = (Matriz*) malloc(sizeof(Matriz));

	mat->n = n;
	mat->m = m;
	mat->cont = (int**) malloc(n * sizeof(int*));
	for(int i = 0; i < n; i++)
		mat->cont[i] = (int*) malloc(m * sizeof(int));

	return mat;
}

void liberarMatriz(Matriz *m)
{
	for(int i = 0; i < m->n; i++)
		free(m->cont[i]);
	free(m->cont);
	free(m);
}

Matriz *gerarMatriz(int n, int m)
{
	Matriz *mat = criarMatriz(n, m);
	
	for(int i = 0; i < n; i++)
		for(int j = 0; j < m; j++)
			{
				mat->cont[i][j] = rand() % 100;
			}

	return mat;
}

void printarMatriz(Matriz *mat)
{
	for(int i = 0; i < mat->n; i++)
	{
		for(int j = 0; j < mat->m; j++)
			printf("%d ", mat->cont[i][j]);
		printf("\n");
	}
}

void multiplicarMatrizes(Matriz *a, Matriz *b, Matriz *c)
{
	for(int i = 0; i < a->n; i++)
		for(int j = 0; j < b->n; j++)
		{
			c->cont[i][j] = 0;
			for(int k = 0; k < b->m; k++)
				c->cont[i][j] += a->cont[i][k] * b->cont[j][k];
		}
}

Matriz *lerMatriz(char *nome, int n, int m, short int trans)
{
	Matriz *mat = NULL;
	FILE *f = fopen(nome, "r");
	if(trans)
	{
		mat = criarMatriz(m, n);

		for(int i = 0; i < n; i++)
			for(int j = 0; j < m; j++)
				fscanf(f, " %d", &(mat->cont[j][i]));
	}
	else
	{
		mat = criarMatriz(n, m);

		for(int i = 0; i < n; i++)
			for(int j = 0; j < m; j++)
				fscanf(f, " %d", &(mat->cont[i][j]));
	}
	fclose(f);

	return mat;
}

void salvarMatriz(Matriz *mat, short int trans)
{
	static int i = 0;

	char nome[100];

	if(trans)	sprintf(nome, "%d-%dx%d.txt", i, mat->m, mat->n);
	else		sprintf(nome, "%d-%dx%d.txt", i, mat->n, mat->m);

	FILE *f = fopen(nome, "w");

	if(trans)
		for(int i = 0; i < mat->m; i++)
		{
			for(int j = 0; j < mat->n; j++)
				fprintf(f, "%d ", mat->cont[j][i]);
			fprintf(f, "\n");
		}
	else
		for(int i = 0; i < mat->n; i++)
		{
			for(int j = 0; j < mat->m; j++)
				fprintf(f, "%d ", mat->cont[i][j]);
			fprintf(f, "\n");
		}

	fclose(f);
	i++;
}

struct _input
{
	Matriz *a;
	Matriz *b;
	Matriz *c;
	short int salvar;
}; typedef struct _input Input;

Input *lerInput(int argc, char **argv)
{
	if(argc >= 6)
	{
		Input *i = (Input *) malloc(sizeof(Input));
		i->salvar = 0;

		int n1, m1, n2, m2;
		char op;

		op = argv[1][0];
		
		sscanf(argv[2], " %d", &n1);
		sscanf(argv[3], " %d", &m1);
		sscanf(argv[4], " %d", &n2);
		sscanf(argv[5], " %d", &m2);
	
		if(m1 == n2)
		{
			Matriz *a, *b, *c;		
			
			switch(op)
			{
				case 'g':
					srand(time(NULL));
					a = gerarMatriz(n1, m1);
					b = gerarMatriz(m2, n2); //INVERTIDOS PARA A TRANSPOSTA
					if(argc == 7 && argv[6][0] == 's')
						i->salvar = 1;
					break;
				case 'f':
					a = lerMatriz(argv[6], n1, m1, NTRANS);
					b = lerMatriz(argv[7], n2, m2, TRANS);
					break;
				default:
					return 0;
			}
			c = criarMatriz(n1, m2);

			i->a = a;
			i->b = b;
			i->c = c;

			return i;
		}
		else
			printf("Matrizes Incompativeis!\n");
	}
	else
		printf("Argumentos invalidos!\n");

	return NULL;
}

double medirTempoExecMul(Input *i)
{
	clock_t tempo = clock();
	multiplicarMatrizes(i->a, i->b, i->c);
	tempo = clock() - tempo;

	return ((double) tempo / CLOCKS_PER_SEC);
}

void salvarELiberarMatrizes(Input *i)
{	
	if(i->salvar)
	{
		salvarMatriz(i->a, NTRANS);
		salvarMatriz(i->b, TRANS);
	}
	salvarMatriz(i->c, NTRANS);

	liberarMatriz(i->a);
	liberarMatriz(i->b);
	liberarMatriz(i->c);
	free(i);
}

int main(int argc, char ** argv)
{
	clock_t tempo = clock();
	Input *i = lerInput(argc, argv);
	printf("Tempo de criacao: %lf\n", (((double) clock() - tempo) / CLOCKS_PER_SEC));
	printf("Tempo de execucao: %lf\n", medirTempoExecMul(i));
	salvarELiberarMatrizes(i);

	return 0;
}
