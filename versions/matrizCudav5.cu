/*
#v1
Ideia: Transformar as matrizes em transpostas para nao precisar fazer ler dois ponteiros, apenas usar o deslocamento
Resultado: Aumento de performance. Tempo 1/8 vezes o anterior #8.2 -> 1.1

#v2
Ideia: Transformar matriz em vetor para preparar para CUDA
Resultado: Perda de desempenho. Tempo 2.4 vezes o anterior #1.1 -> 2.4
	#v2.1
	Ideia: otimizar o codigo antes do CUDA procurando por calculos repetidos e os atribuindo a auxiliares
	Resultado: Ganho de desempenho. Tempo 10/15 vezes o anterior #2.4 -> 1.55

#v3
Ideia: Utilizar a GPU para fazer os calculos quando a matriz for grande
Resultado: Ganho de desempenho. Tempo 10/50 vezes o anterior #1.55 -> 0.35

#v4 - DESCONSIDERADO
Otimizar processamento na CPU com e SSE
Resultado: N/A

#v4.1
Utilizar AVX ao inves de SSE
Resultado: Ganho de desempenho. Tempo 10/28 vezes o anterior # 1.55 -> 0.55
	#v4.2
	Encontradas novas contas frequentes e foram trocadas para variaveis auxiliar

#v5 - DESCONSIDERADO
Utilizar GPU e CPU ao mesmo tempo e remover if do kernel da GPU
Resultado: Perda de desempenho

#v5.1
Utilizar OpenMp para paralelizar codigos na CPU
Resultado: Ganho de performance. Tempo > 1/2 vezes o anterior #0.55 -> 0.22 ######AINDA NAO ESTIMADO
	#5.1.1
	Melhora pequena, porem consideravel ao guardar o endereco de C[indiceC] 
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <immintrin.h>
#include <omp.h>

#define NTRANS 0
#define TRANS 1

#define OPMAX 1024
#define MAXTHREADS_GPU 32	//32*32=1024, Nao se deve ter mais que 1024 threads por bloco

#define MAXTHREADS_CPU 8

#define AVXJUMP 8

struct _matriz
{
	int n;
	int m;
	int *cont;
}; typedef struct _matriz Matriz;

struct _input
{
	Matriz *a;
	Matriz *b;
	Matriz *c;
	short int salvar;
}; typedef struct _input Input;

Matriz *criarMatriz(int n, int m)
{
	Matriz *mat = (Matriz*) malloc(sizeof(Matriz));

	mat->n = n;
	mat->m = m;
	mat->cont = (int*) malloc(n * m * sizeof(int*));

	return mat;
}

void liberarMatriz(Matriz *m)
{
	free(m->cont);
	free(m);
}

Matriz *gerarMatriz(int n, int m)
{
	Matriz *mat = criarMatriz(n, m);
	
	for(int i = 0; i < n; i++)
		for(int j = 0; j < m; j++)
			{
				mat->cont[i * m + j] = rand() % 100;
			}

	return mat;
}

void printarMatriz(Matriz *mat)
{
	for(int i = 0; i < mat->n; i++)
	{
		for(int j = 0; j < mat->m; j++)
			printf("%d ", mat->cont[i * mat->m + j]);
		printf("\n");
	}
}

/*void multiplicarMatrizes(Matriz *a, Matriz *b, Matriz *c)
{
	int aux;
	for(int i = 0; i < a->n; i++)
		for(int j = 0; j < b->n; j++)
		{
			aux = i * c->m + j;
			c->cont[aux] = 0;
			for(int k = 0; k < b->m; k++)
				c->cont[aux] += a->cont[i * a->m + k] * b->cont[j * b->m + k];
		}
}*/

void multiplicarMatrizesAVX(Matriz *matA, Matriz *matB, Matriz *matC)
{
	int *a = matA->cont, *b = matB->cont, *c = matC->cont, *alvoC;
	int aN = matA->n, bN = matB->n, aM = matA->m, bM = matB->m, cM = matC->m;

	__m256i mask = _mm256_setr_epi32(-1, -2, -3, -4, -5, -6, -7, -8);	//MASCARA INFORMA QUE SERAO USADOS OS 256 BITS DO AVX
	__m256i regMults;

	int indiceA, indiceB;
	int limK = bM - (bM % AVXJUMP);

	int i, j, k;

	#pragma omp parallel for firstprivate(aN, indiceA, aM, bN, indiceB, bM, alvoC, c, cM, limK, a, b) private(i, j, k, regMults)
	for(i = 0; i < aN; i++)
	{
		indiceA = i * aM;
		for(j = 0; j < bN; j++)
		{
			indiceB = j * bM;
			alvoC = &c[i * cM + j];
			*alvoC = 0;

			for(k = 0; k < limK; k += AVXJUMP)	//LOOP PARA MULTIPLOS DE 8, LIMITE DO AVX E THREADS
			{
					regMults = _mm256_mullo_epi32(_mm256_maskload_epi32(&a[indiceA + k], mask),
									_mm256_maskload_epi32(&b[indiceB + k], mask));
					regMults = _mm256_hadd_epi32(regMults, regMults);
					regMults = _mm256_hadd_epi32(regMults, regMults);
					*alvoC += (_mm256_extract_epi32(regMults, 0) + _mm256_extract_epi32(regMults, 7));
			}
		}
	}

	if(limK < bM)	//SEPARADO DO LOOP J PRINCIPAL PARA EVITAR CALCULOS EM MULTIPLOS DE 8
	{
		#pragma omp parallel for firstprivate(aN, indiceA, aM, bN, indiceB, bM, alvoC, c, cM, limK, a, b) private(i, j, k)
		for(i = 0; i < aN; i++)
		{
			indiceA = i * aM;
			for(j = 0; j < bN; j++)
			{
				indiceB = j * bM;
				alvoC = &c[i * cM + j];

				for(k = limK; k < bM; k++)	//LOOP PARA CASO O N°COL DE C NAO SEJA MULTIPLO DE 8
					*alvoC += a[indiceA + k] * b[indiceB + k];
			}
		}

	}
}

//KERNEL QUE EXECUTA AS INSTRUCOES NA GPU
__global__ void kernelMulMatriz(int *a, int aN, int aM, int *b, int bN, int bM, int *c, int cM)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if(i < aN && j < bN)
	{
		int indiceA = i * aM;
		int indiceB = j * bM;
		int *alvoC = &c[i * cM + j];
		*alvoC = 0;
		for(int k = 0; k < bM; k++)
			*alvoC += a[indiceA + k] * b[indiceB + k];
	}
}

//FUNCAO QUE PREPARA PARA A GPU
void multiplicarMatrizesCUDA(Matriz *a, Matriz *b, Matriz *c)
{
	int *d_a, *d_b, *d_c, opCount = a->n * b->n;
	dim3 blocksPerGrid(1, 1), threadsPerBlock(a->n, b->n);
	if(opCount > OPMAX)
	{
		threadsPerBlock.x = MAXTHREADS_GPU;
		threadsPerBlock.y = MAXTHREADS_GPU;
		blocksPerGrid.x = ceil(((double) a->n / MAXTHREADS_GPU));
		blocksPerGrid.y = ceil(((double) b->n / MAXTHREADS_GPU));
	}

	hipMalloc(&d_a, sizeof(int) * a->n * a->m);
	hipMalloc(&d_b, sizeof(int) * b->n * b->m);
	hipMalloc(&d_c, sizeof(int) * c->n * c->m);
	hipMemcpy(d_a, a->cont, sizeof(int) * a->n * a->m, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b->cont, sizeof(int) * b->n * b->m, hipMemcpyHostToDevice);

	kernelMulMatriz <<<blocksPerGrid, threadsPerBlock>>> (d_a, a->n, a->m, d_b, b->n, b->m, d_c, c->m);

	hipMemcpy(c->cont, d_c, sizeof(int) * c->n * c->m, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	hipDeviceSynchronize();
}

Matriz *lerMatriz(char *nome, int n, int m, short int trans)
{
	Matriz *mat = NULL;
	FILE *f = fopen(nome, "r");
	if(trans)
	{
		mat = criarMatriz(m, n);

		for(int i = 0; i < n; i++)
			for(int j = 0; j < m; j++)
				fscanf(f, " %d", &(mat->cont[j * n + i]));
	}
	else
	{
		mat = criarMatriz(n, m);

		for(int i = 0; i < n; i++)
			for(int j = 0; j < m; j++)
				fscanf(f, " %d", &(mat->cont[i * m + j]));
	}
	fclose(f);

	return mat;
}

void salvarMatriz(Matriz *mat, short int trans)
{
	static int i = 0;
	char nome[100];

	if(trans)	sprintf(nome, "%d-%dx%d.txt", i, mat->m, mat->n);
	else		sprintf(nome, "%d-%dx%d.txt", i, mat->n, mat->m);

	FILE *f = fopen(nome, "w");

	if(trans)
		for(int i = 0; i < mat->m; i++)
		{
			for(int j = 0; j < mat->n; j++)
				fprintf(f, "%d ", mat->cont[j * mat->m + i]);
			fprintf(f, "\n");
		}
	else
		for(int i = 0; i < mat->n; i++)
		{
			for(int j = 0; j < mat->m; j++)
				fprintf(f, "%d ", mat->cont[i * mat->m + j]);
			fprintf(f, "\n");
		}

	fclose(f);
	i++;
}

Input *lerInput(int argc, char **argv)
{
	if(argc >= 6)
	{
		Input *i = (Input *) malloc(sizeof(Input));
		i->salvar = 0;

		int n1, m1, n2, m2;
		char op;

		op = argv[1][0];
		
		sscanf(argv[2], " %d", &n1);
		sscanf(argv[3], " %d", &m1);
		sscanf(argv[4], " %d", &n2);
		sscanf(argv[5], " %d", &m2);
	
		if(m1 == n2)
		{
			Matriz *a, *b, *c;		
			
			switch(op)
			{
				case 'g':
					srand(time(NULL));
					a = gerarMatriz(n1, m1);
					b = gerarMatriz(m2, n2); //INVERTIDOS PARA A TRANSPOSTA
					if(argc == 7 && argv[6][0] == 's')
						i->salvar = 1;
					break;
				case 'f':
					a = lerMatriz(argv[6], n1, m1, NTRANS);
					b = lerMatriz(argv[7], n2, m2, TRANS);
					break;
				default:
					return 0;
			}
			c = criarMatriz(n1, m2);

			i->a = a;
			i->b = b;
			i->c = c;

			return i;
		}
		else
			printf("Matrizes Incompativeis!\n");
	}
	else
		printf("Argumentos invalidos!\n");

	return NULL;
}

double medirTempoExecMul(Input *i, void mul(Matriz*, Matriz*, Matriz*))
{
	clock_t tempo = clock();
	mul(i->a, i->b, i->c);
	tempo = clock() - tempo;

	return ((double) tempo / CLOCKS_PER_SEC);
}

void salvarELiberarMatrizes(Input *i)
{	
	if(i->salvar)
	{
		salvarMatriz(i->a, NTRANS);
		salvarMatriz(i->b, TRANS);
	}
	salvarMatriz(i->c, NTRANS);

	liberarMatriz(i->a);
	liberarMatriz(i->b);
	liberarMatriz(i->c);
	free(i);
}

void inicializar()
{
	omp_set_num_threads(MAXTHREADS_CPU);
}

int main(int argc, char ** argv)
{
	inicializar();
	clock_t tempo = clock();
	Input *i = lerInput(argc, argv);
	printf("Tempo de criacao: %lf\n", (((double) clock() - tempo) / CLOCKS_PER_SEC));
	printf("Tempo de execucao: %lf\n", medirTempoExecMul(i, &multiplicarMatrizesCUDA));
	salvarELiberarMatrizes(i);

	return 0;
}
